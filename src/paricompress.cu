#include "hip/hip_runtime.h"
#ifdef _WIN32
#include <windows.h>
#endif
#include <GL/gl.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/transform_scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include "paricompress.h"

static double _compute_time;
static double _mem_transfer_time;
static double _total_time;

__host__ __device__ static void extractTile4x4(uint32_t offset, const uint8_t *pixels, int width, uint8_t out_tile[64]);
__host__ __device__ static void getMinMaxColors(uint8_t tile[64], uint8_t color_min[3], uint8_t color_max[3]);
__host__ __device__ static uint16_t colorTo565(uint8_t color[3]);
__host__ __device__ static uint32_t colorDistance(uint8_t tile[64], int t_offset, uint8_t colors[16], int c_offset);
__host__ __device__ static uint32_t colorIndices(uint8_t tile[64], uint8_t color_min[3], uint8_t color_max[3]);
__host__ __device__ static void writeUint16(uint8_t *buffer, uint32_t offset, uint16_t value);
__host__ __device__ static void writeUint32(uint8_t *buffer, uint32_t offset, uint32_t value);

__device__ static void extractCGTile4x4(uint32_t offset_x, uint32_t offset_y, const hipSurfaceObject_t pixels, uint8_t out_tile[64]);

static uint64_t currentTime();


// CUDA Thrust transformer to change data type
template<typename T1, typename T2>
struct typecast
{
    __host__ __device__ T2 operator()(const T1 &x) const
    {
        return static_cast<T2>(x);
    }
};

// CUDA Thrust functors
struct PariGrayscaleFunctor
{
    const uint8_t *rgba;
    uint8_t *gray;
    size_t size;       
    PariGrayscaleFunctor(thrust::device_vector<uint8_t> const& rgba_input, thrust::device_vector<uint8_t>& gray_output)
    {
        rgba = thrust::raw_pointer_cast(rgba_input.data());
        gray = thrust::raw_pointer_cast(gray_output.data());
        size = rgba_input.size() / 4;
    } 
    __host__ __device__	void operator()(int thread_id)
    {
        if (thread_id < size)
        {
            float red = (float)rgba[4 * thread_id + 0];
            float green = (float)rgba[4 * thread_id + 1];
            float blue = (float)rgba[4 * thread_id + 2];
            gray[thread_id] = (uint8_t)(0.299f * red + 0.587f * green + 0.114f * blue);
        }
    }
};

struct PariDxt1Functor
{
    const uint8_t *rgba;
    uint8_t *dxt1;
    uint32_t width;
    size_t size;
    PariDxt1Functor(thrust::device_vector<uint8_t> const& rgba_input, thrust::device_vector<uint8_t>& dxt1_output, uint32_t width_input)
    {
        rgba = thrust::raw_pointer_cast(rgba_input.data());
        dxt1 = thrust::raw_pointer_cast(dxt1_output.data());
        width = width_input;
        size = rgba_input.size() / 64;
    }
    __host__ __device__ void operator()(int thread_id)
    {
        if (thread_id < size)
        {
            uint8_t tile[64];
            uint8_t color_min[3];
            uint8_t color_max[3];

            // px_ (x and y pixel indices)
            // tile_ (x and y tile indices)
      	    uint32_t tile_x = thread_id % (width / 4);
            uint32_t tile_y = thread_id / (width / 4);
            uint32_t px_x = tile_x * 4;
            uint32_t px_y = tile_y * 4;

            uint32_t offset = (px_y * width * 4) + (px_x * 4);
            uint32_t write_pos = (tile_y * (width / 4) * 8) + (tile_x * 8);

            extractTile4x4(offset, rgba, width, tile);
            getMinMaxColors(tile, color_min, color_max);
            writeUint16(dxt1, write_pos, colorTo565(color_max));
       	    writeUint16(dxt1, write_pos + 2, colorTo565(color_min));
       	    writeUint32(dxt1, write_pos + 4, colorIndices(tile, color_min, color_max));
        }
    }
};

struct PariActivePixelFunctor
{
    const uint8_t *rgba;
    const float *depth;
    uint8_t *new_run;
    uint8_t *is_active;
    int width;
    int height;
    float max_depth;
    PariActivePixelFunctor(int width_input, int height_input, thrust::device_vector<uint8_t> const& rgba_input,
                           thrust::device_vector<float> const& depth_input, thrust::device_vector<uint8_t>& new_run_output,
                           thrust::device_vector<uint8_t>& is_active_output)
    {
        rgba = thrust::raw_pointer_cast(rgba_input.data());
        depth = thrust::raw_pointer_cast(depth_input.data());
        new_run = thrust::raw_pointer_cast(new_run_output.data());
        is_active = thrust::raw_pointer_cast(is_active_output.data());
        width = width_input;
        height = height_input;
        max_depth = 1.0f;
    }
    __host__ __device__ void operator()(int thread_id)
    {
        if (thread_id < width * height)
        {
            // whether or not pixel is active
            is_active[thread_id] = (uint8_t)(depth[thread_id] != max_depth);
            
            // whether or not pixel starts a new run
            if (thread_id == 0)
            {
                new_run[thread_id] = 1;
            }
            else
            {
                uint8_t prev_active = (uint8_t)(depth[thread_id - 1] != max_depth);
                new_run[thread_id] = (uint8_t)(is_active[thread_id] != prev_active);
            }
        }
    }
};

struct PariActivePixelFinalizeFunctor
{
    const uint8_t *rgba;
    const float *depth;
    uint8_t *is_active;
    uint8_t *new_run;
    uint32_t *run_index;
    uint32_t *run_length;
    uint32_t *active_index;
    int width;
    int height;
    uint8_t *compressed;
    uint32_t *compressed_size;
    float max_depth;
    PariActivePixelFinalizeFunctor(int width_input, int height_input, thrust::device_vector<uint8_t> const& rgba_input,
                                   thrust::device_vector<float> const& depth_input, thrust::device_vector<uint8_t>& is_active_input,
                                   thrust::device_vector<uint8_t>& new_run_input, thrust::device_vector<uint32_t>& run_idx_input,
                                   thrust::device_vector<uint32_t>& run_length_input, thrust::device_vector<uint32_t>& active_idx_input,
                                   thrust::device_vector<uint8_t>& output, thrust::device_vector<uint32_t>& output_size)
    {
        rgba = thrust::raw_pointer_cast(rgba_input.data());
        depth = thrust::raw_pointer_cast(depth_input.data());
        is_active = thrust::raw_pointer_cast(is_active_input.data());
        new_run = thrust::raw_pointer_cast(new_run_input.data());
        run_index = thrust::raw_pointer_cast(run_idx_input.data());
        run_length = thrust::raw_pointer_cast(run_length_input.data());
        active_index = thrust::raw_pointer_cast(active_idx_input.data());
        compressed = thrust::raw_pointer_cast(output.data());
        compressed_size = thrust::raw_pointer_cast(output_size.data());
        width = width_input;
        height = height_input;
        max_depth = 1.0f;
    }
    __host__ __device__ void operator()(int thread_id)
    {
        if(thread_id < width * height && is_active[thread_id]) // active pixels only
        {
            uint32_t write_pos = 8 * (active_index[thread_id] + ((run_index[thread_id] - 1) / 2) + 1);
            
            memcpy(compressed + write_pos, rgba + (4 * thread_id), 4);
            memcpy(compressed + write_pos + 4, depth + thread_id, 4);
            if (new_run[thread_id] == 1)
            {
                uint32_t num_inactive = (run_index[thread_id] > 1) ? run_length[run_index[thread_id] - 2] : 0;
                uint32_t num_active = run_length[run_index[thread_id] - 1];
                memcpy(compressed + write_pos - 8, &num_inactive, 4);
                memcpy(compressed + write_pos - 4, &num_active, 4);
            }
        }
        if (thread_id == (width * height) - 1) // final pixel - write compressed size
        {
            uint32_t active_run = run_index[thread_id] + is_active[thread_id] - 2;
            uint32_t write_pos = 8 * (active_index[thread_id] + (active_run / 2) + 1);
        
            compressed_size[0] = write_pos + 8;
            if (is_active[thread_id] == 0)
            {
                uint32_t num_inactive = run_length[run_index[thread_id] - 1];
                uint32_t num_active = 0;
                memcpy(compressed + write_pos, &num_inactive, 4);
                memcpy(compressed + write_pos + 4, &num_active, 4);
            }
        }
    }
};

struct PariCGGrayscaleFunctor
{
    hipSurfaceObject_t rgba;
    uint8_t *gray;
    uint32_t width;
    uint32_t height;
    PariCGGrayscaleFunctor(hipSurfaceObject_t const& rgba_input, thrust::device_vector<uint8_t>& gray_output,
                           uint32_t width_input, uint32_t height_input)
    {
        rgba = rgba_input;
        gray = thrust::raw_pointer_cast(gray_output.data());
        width = width_input;
        height = height_input;
    } 
    __device__	void operator()(int thread_id)
    {
        if (thread_id < (width * height))
        {  
            uchar4 color;
            surf2Dread(&color, rgba, 4 * (thread_id % width), thread_id / width);
            gray[thread_id] = (uint8_t)(0.299f * color.x + 0.587f * color.y + 0.114f * color.z);
        }
    }
};

struct PariCGDxt1Functor
{
    hipSurfaceObject_t rgba;
    uint8_t *dxt1;
    uint32_t width;
    uint32_t height;
    PariCGDxt1Functor(hipSurfaceObject_t const& rgba_input, thrust::device_vector<uint8_t>& dxt1_output,
                      uint32_t width_input, uint32_t height_input)
    {
        rgba = rgba_input;
        dxt1 = thrust::raw_pointer_cast(dxt1_output.data());
        width = width_input;
        height = height_input;
    }
    __device__ void operator()(int thread_id)
    {
        if (thread_id < (width * height / 16))
        {
            uint8_t tile[64];
            uint8_t color_min[3];
            uint8_t color_max[3];

            // px_ (x and y pixel indices)
            // tile_ (x and y tile indices)
      	    uint32_t tile_x = thread_id % (width / 4);
            uint32_t tile_y = thread_id / (width / 4);
            uint32_t px_x = tile_x * 4;
            uint32_t px_y = tile_y * 4;

            uint32_t write_pos = (tile_y * (width / 4) * 8) + (tile_x * 8);

            extractCGTile4x4(px_x, px_y, rgba, tile);
            getMinMaxColors(tile, color_min, color_max);
            writeUint16(dxt1, write_pos, colorTo565(color_max));
       	    writeUint16(dxt1, write_pos + 2, colorTo565(color_min));
       	    writeUint32(dxt1, write_pos + 4, colorIndices(tile, color_min, color_max));
        }
    }
};

struct PariCGActivePixelFunctor
{
    hipSurfaceObject_t depth;
    uint8_t *new_run;
    uint8_t *is_active;
    int width;
    int height;
    float max_depth;
    PariCGActivePixelFunctor(hipSurfaceObject_t const& depth_input, thrust::device_vector<uint8_t>& new_run_output,
                             thrust::device_vector<uint8_t>& is_active_output, int width_input, int height_input)
    {
        depth = depth_input;
        new_run = thrust::raw_pointer_cast(new_run_output.data());
        is_active = thrust::raw_pointer_cast(is_active_output.data());
        width = width_input;
        height = height_input;
        max_depth = 1.0f;
    }
    __device__ void operator()(int thread_id)
    {
        if (thread_id < width * height)
        {
            float px_depth;
            surf2Dread(&px_depth, depth, 4 * (thread_id % width), thread_id / width);
            
            // whether or not pixel is active
            is_active[thread_id] = (uint8_t)(px_depth != max_depth);
            
            // whether or not pixel starts a new run
            if (thread_id == 0)
            {
                new_run[thread_id] = 1;
            }
            else
            {
                float prev_depth;
                surf2Dread(&prev_depth, depth, 4 * ((thread_id - 1) % width), (thread_id - 1) / width);
                
                uint8_t prev_active = (uint8_t)(prev_depth != max_depth);
                new_run[thread_id] = (uint8_t)(is_active[thread_id] != prev_active);
            }
        }
    }
};

struct PariCGActivePixelFinalizeFunctor
{
    hipSurfaceObject_t rgba;
    hipSurfaceObject_t depth;
    uint8_t *is_active;
    uint8_t *new_run;
    uint32_t *run_index;
    uint32_t *run_length;
    uint32_t *active_index;
    int width;
    int height;
    uint8_t *compressed;
    uint32_t *compressed_size;
    float max_depth;
    PariCGActivePixelFinalizeFunctor(hipSurfaceObject_t const& rgba_input, hipSurfaceObject_t const& depth_input,
                                     thrust::device_vector<uint8_t>& is_active_input, thrust::device_vector<uint8_t>& new_run_input,
                                     thrust::device_vector<uint32_t>& run_idx_input, thrust::device_vector<uint32_t>& run_length_input,
                                     thrust::device_vector<uint32_t>& active_idx_input, thrust::device_vector<uint8_t>& output,
                                     thrust::device_vector<uint32_t>& output_size, int width_input, int height_input)
    {
        rgba = rgba_input;
        depth = depth_input;
        is_active = thrust::raw_pointer_cast(is_active_input.data());
        new_run = thrust::raw_pointer_cast(new_run_input.data());
        run_index = thrust::raw_pointer_cast(run_idx_input.data());
        run_length = thrust::raw_pointer_cast(run_length_input.data());
        active_index = thrust::raw_pointer_cast(active_idx_input.data());
        compressed = thrust::raw_pointer_cast(output.data());
        compressed_size = thrust::raw_pointer_cast(output_size.data());
        width = width_input;
        height = height_input;
        max_depth = 1.0f;
    }
    __device__ void operator()(int thread_id)
    {
        if(thread_id < width * height && is_active[thread_id]) // active pixels only
        {
            uint32_t write_pos = 8 * (active_index[thread_id] + ((run_index[thread_id] - 1) / 2) + 1);
            
            uchar4 px_color;
            float px_depth;
            surf2Dread(&px_color, rgba, 4 * (thread_id % width), thread_id / width);
            surf2Dread(&px_depth, depth, 4 * (thread_id % width), thread_id / width);
            memcpy(compressed + write_pos, &px_color, 4);
            memcpy(compressed + write_pos + 4, &px_depth, 4);
            if (new_run[thread_id] == 1)
            {
                uint32_t num_inactive = (run_index[thread_id] > 1) ? run_length[run_index[thread_id] - 2] : 0;
                uint32_t num_active = run_length[run_index[thread_id] - 1];
                memcpy(compressed + write_pos - 8, &num_inactive, 4);
                memcpy(compressed + write_pos - 4, &num_active, 4);
            }
        }
        if (thread_id == (width * height) - 1) // final pixel - write compressed size
        {
            uint32_t active_run = run_index[thread_id] + is_active[thread_id] - 2;
            uint32_t write_pos = 8 * (active_index[thread_id] + (active_run / 2) + 1);
        
            compressed_size[0] = write_pos + 8;
            if (is_active[thread_id] == 0)
            {
                uint32_t num_inactive = run_length[run_index[thread_id] - 1];
                uint32_t num_active = 0;
                memcpy(compressed + write_pos, &num_inactive, 4);
                memcpy(compressed + write_pos + 4, &num_active, 4);
            }
        }
    }
};

struct PariCGSubActivePixelFunctor
{
    hipSurfaceObject_t depth;
    uint8_t *new_run;
    uint8_t *is_active;
    int texture_width;
    int texture_height;
    int texture_viewport_x;
    int texture_viewport_y;
    int texture_viewport_w;
    int texture_viewport_h;
    int ap_width;
    int ap_height;
    int ap_viewport_x;
    int ap_viewport_y;
    int ap_viewport_w;
    int ap_viewport_h;
    float max_depth;
    PariCGSubActivePixelFunctor(hipSurfaceObject_t const& depth_input, thrust::device_vector<uint8_t>& new_run_output,
                             thrust::device_vector<uint8_t>& is_active_output, int texture_width_input, int texture_height_input,
                             int *texture_viewport_input, int ap_width_input, int ap_height_input, int *ap_viewport_input)
    {
        depth = depth_input;
        new_run = thrust::raw_pointer_cast(new_run_output.data());
        is_active = thrust::raw_pointer_cast(is_active_output.data());
        texture_width = texture_width_input;
        texture_height = texture_height_input;
        texture_viewport_x = texture_viewport_input[0];
        texture_viewport_y = texture_viewport_input[1];
        texture_viewport_w = texture_viewport_input[2];
        texture_viewport_h = texture_viewport_input[3];
        ap_width = ap_width_input;
        ap_height = ap_height_input;
        ap_viewport_x = ap_viewport_input[0];
        ap_viewport_y = ap_viewport_input[1];
        ap_viewport_w = ap_viewport_input[2];
        ap_viewport_h = ap_viewport_input[3];
        max_depth = 1.0f;
    }
    __device__ void operator()(int thread_id)
    {
        if (thread_id < ap_width * ap_height)
        {
            // whether or not pixel is active
            is_active[thread_id] = isActive(thread_id);
        
            // whether or not pixel starts a new run
            if (thread_id == 0)
            {
                new_run[thread_id] = 1;
            }
            else
            {
                uint8_t prev_active = isActive(thread_id - 1);
                new_run[thread_id] = (uint8_t)(is_active[thread_id] != prev_active);
            }
        }
    }
    __device__ uint8_t isActive(int thread_id)
    {
        uint8_t active = 0;
        int px_x = thread_id % ap_width;
        int px_y = thread_id / ap_width;
        
        // pixel inside viewport
        if (px_x >= ap_viewport_x && px_x < (ap_viewport_x + ap_viewport_w) &&
            px_y >= ap_viewport_y && px_y < (ap_viewport_y + ap_viewport_h))
        {
            int px_texture_x = px_x - ap_viewport_x + texture_viewport_x;
            int px_texture_y = px_y - ap_viewport_y + texture_viewport_y;
            
            float px_depth;
            surf2Dread(&px_depth, depth, 4 * px_texture_x, px_texture_y);
            
            active = (uint8_t)(px_depth != max_depth);
        }
        return active;
    }
};

struct PariCGSubActivePixelFinalizeFunctor
{
    hipSurfaceObject_t rgba;
    hipSurfaceObject_t depth;
    uint8_t *is_active;
    uint8_t *new_run;
    uint32_t *run_index;
    uint32_t *run_length;
    uint32_t *active_index;
    int texture_width;
    int texture_height;
    int texture_viewport_x;
    int texture_viewport_y;
    int texture_viewport_w;
    int texture_viewport_h;
    int ap_width;
    int ap_height;
    int ap_viewport_x;
    int ap_viewport_y;
    int ap_viewport_w;
    int ap_viewport_h;
    uint8_t *compressed;
    uint32_t *compressed_size;
    float max_depth;
    PariCGSubActivePixelFinalizeFunctor(hipSurfaceObject_t const& rgba_input, hipSurfaceObject_t const& depth_input,
                                        thrust::device_vector<uint8_t>& is_active_input, thrust::device_vector<uint8_t>& new_run_input,
                                        thrust::device_vector<uint32_t>& run_idx_input, thrust::device_vector<uint32_t>& run_length_input,
                                        thrust::device_vector<uint32_t>& active_idx_input, thrust::device_vector<uint8_t>& output,
                                        thrust::device_vector<uint32_t>& output_size, int texture_width_input, int texture_height_input,
                                        int *texture_viewport_input, int ap_width_input, int ap_height_input, int *ap_viewport_input)
    {
        rgba = rgba_input;
        depth = depth_input;
        is_active = thrust::raw_pointer_cast(is_active_input.data());
        new_run = thrust::raw_pointer_cast(new_run_input.data());
        run_index = thrust::raw_pointer_cast(run_idx_input.data());
        run_length = thrust::raw_pointer_cast(run_length_input.data());
        active_index = thrust::raw_pointer_cast(active_idx_input.data());
        compressed = thrust::raw_pointer_cast(output.data());
        compressed_size = thrust::raw_pointer_cast(output_size.data());
        texture_width = texture_width_input;
        texture_height = texture_height_input;
        texture_viewport_x = texture_viewport_input[0];
        texture_viewport_y = texture_viewport_input[1];
        texture_viewport_w = texture_viewport_input[2];
        texture_viewport_h = texture_viewport_input[3];
        ap_width = ap_width_input;
        ap_height = ap_height_input;
        ap_viewport_x = ap_viewport_input[0];
        ap_viewport_y = ap_viewport_input[1];
        ap_viewport_w = ap_viewport_input[2];
        ap_viewport_h = ap_viewport_input[3];
        max_depth = 1.0f;
    }
    __device__ void operator()(int thread_id)
    {
        if(thread_id < ap_width * ap_height && is_active[thread_id]) // active pixels only
        {
            uint32_t write_pos = 8 * (active_index[thread_id] + ((run_index[thread_id] - 1) / 2) + 1);
            
            int px_texture_x = (thread_id % ap_width) - ap_viewport_x + texture_viewport_x;
            int px_texture_y = (thread_id / ap_width) - ap_viewport_y + texture_viewport_y;
            
            uchar4 px_color;
            float px_depth;
            surf2Dread(&px_color, rgba, 4 * px_texture_x, px_texture_y);
            surf2Dread(&px_depth, depth, 4 * px_texture_x, px_texture_y);
            memcpy(compressed + write_pos, &px_color, 4);
            memcpy(compressed + write_pos + 4, &px_depth, 4);
            if (new_run[thread_id] == 1)
            {
                uint32_t num_inactive = (run_index[thread_id] > 1) ? run_length[run_index[thread_id] - 2] : 0;
                uint32_t num_active = run_length[run_index[thread_id] - 1];
                memcpy(compressed + write_pos - 8, &num_inactive, 4);
                memcpy(compressed + write_pos - 4, &num_active, 4);
            }
        }
        if (thread_id == (ap_width * ap_height) - 1) // final pixel - write compressed size
        {
            uint32_t active_run = run_index[thread_id] + is_active[thread_id] - 2;
            uint32_t write_pos = 8 * (active_index[thread_id] + (active_run / 2) + 1);
        
            compressed_size[0] = write_pos + 8;
            if (is_active[thread_id] == 0)
            {
                uint32_t num_inactive = run_length[run_index[thread_id] - 1];
                uint32_t num_active = 0;
                memcpy(compressed + write_pos, &num_inactive, 4);
                memcpy(compressed + write_pos + 4, &num_active, 4);
            }
        }
    }
};


// Standard PARI functions
PARI_DLLEXPORT void pariSetGpuDevice(int device)
{
    if (device == PARI_DEVICE_OPENGL)
    {
        unsigned int device_count;
        int devices[8];
        hipGLGetDevices(&device_count, devices, 8, hipGLDeviceListAll);
        device = devices[0];
    }
    hipSetDevice(device);
}

PARI_DLLEXPORT PariGpuBuffer pariAllocateGpuBuffer(uint32_t width, uint32_t height, PariEnum type)
{
    PariGpuBuffer buffers;
    switch (type)
    {
        case PARI_IMAGE_RGBA:
            buffers = (PariGpuBuffer)malloc(sizeof(void*));
            buffers[0] = (void*)(new thrust::device_vector<uint8_t>(width * height * 4));
            break;
        case PARI_IMAGE_DEPTH32F:
            buffers = (PariGpuBuffer)malloc(sizeof(void*));
            buffers[0] = (void*)(new thrust::device_vector<float>(width * height));
            break;
        case PARI_IMAGE_GRAYSCALE:
            buffers = (PariGpuBuffer)malloc(sizeof(void*));
            buffers[0] = (void*)(new thrust::device_vector<uint8_t>(width * height));
            break;
        case PARI_IMAGE_RGB:
            buffers = (PariGpuBuffer)malloc(sizeof(void*));
            buffers[0] = (void*)(new thrust::device_vector<uint8_t>(width * height * 3));
            break;
        case PARI_IMAGE_DXT1:
            if (width % 4 != 0 || height % 4 != 0)
            {
                buffers = NULL;
            }
            else
            {
                buffers = (PariGpuBuffer)malloc(sizeof(void*));
                buffers[0] = (void*)(new thrust::device_vector<uint8_t>(width * height / 2));
            }
            break;
        case PARI_IMAGE_ACTIVE_PIXEL:
            buffers = (PariGpuBuffer)malloc(7 * sizeof(void*));
            buffers[0] = (void*)(new thrust::device_vector<uint8_t>(width * height));         // whether or not each pixel starts a new run (0 or 1)
            buffers[1] = (void*)(new thrust::device_vector<uint8_t>(width * height));         // whether or not each pixel is active (0 or 1)
            buffers[2] = (void*)(new thrust::device_vector<uint32_t>(width * height));        // id for each run (inclusive scan of buffers[0])
            buffers[3] = (void*)(new thrust::device_vector<uint32_t>(width * height));        // number of pixels in each run (reduce_by_key of buffers[2])
            buffers[4] = (void*)(new thrust::device_vector<uint32_t>(width * height));        // number of active pixels prior to each pixel (exclusive scan of buffers[1])
            buffers[5] = (void*)(new thrust::device_vector<uint8_t>(width * height * 8 + 8)); // final compressed image
            buffers[6] = (void*)(new thrust::device_vector<uint32_t>(1));                     // size in bytes of final compressed image
            break;
        default:
            buffers = NULL;
            break;
    }
    return buffers;
}

PARI_DLLEXPORT void pariFreeGpuBuffer(PariGpuBuffer buffer, PariEnum type)
{
    switch (type)
    {
        case PARI_IMAGE_RGBA:
            {
                thrust::device_vector<uint8_t> *rgba = (thrust::device_vector<uint8_t>*)buffer[0];
                rgba->clear();
                delete rgba;
            }
            break;
        case PARI_IMAGE_DEPTH32F:
            {
                thrust::device_vector<float> *depth = (thrust::device_vector<float>*)buffer[0];
                depth->clear();
                delete depth;
            }
            break;
        case PARI_IMAGE_GRAYSCALE:
            {
                thrust::device_vector<uint8_t> *gray = (thrust::device_vector<uint8_t>*)buffer[0];
                gray->clear();
                delete gray;
            }
            break;
        case PARI_IMAGE_RGB:
            {
                thrust::device_vector<uint8_t> *rgb = (thrust::device_vector<uint8_t>*)buffer[0];
                rgb->clear();
                delete rgb;
            }
            break;
        case PARI_IMAGE_DXT1:
            {
                thrust::device_vector<uint8_t> *dxt1 = (thrust::device_vector<uint8_t>*)buffer[0];
                dxt1->clear();
                delete dxt1;
            }
            break;
        case PARI_IMAGE_ACTIVE_PIXEL:
            {
                thrust::device_vector<uint8_t> *new_run = (thrust::device_vector<uint8_t>*)buffer[0];
                thrust::device_vector<uint8_t> *is_active = (thrust::device_vector<uint8_t>*)buffer[1];
                thrust::device_vector<uint32_t> *run_id = (thrust::device_vector<uint32_t>*)buffer[2];
                thrust::device_vector<uint32_t> *run_counts = (thrust::device_vector<uint32_t>*)buffer[3];
                thrust::device_vector<uint32_t> *active_idx = (thrust::device_vector<uint32_t>*)buffer[4];
                thrust::device_vector<uint8_t> *ap_image = (thrust::device_vector<uint8_t>*)buffer[5];
                thrust::device_vector<uint32_t> *ap_size = (thrust::device_vector<uint32_t>*)buffer[6];
                new_run->clear();
                is_active->clear();
                run_id->clear();
                run_counts->clear();
                active_idx->clear();
                ap_image->clear();
                ap_size->clear();
                delete new_run;
                delete is_active;
                delete run_id;
                delete run_counts;
                delete active_idx;
                delete ap_image;
                delete ap_size;
            }
            break;
        default:
            break;
    }
}

PARI_DLLEXPORT void pariRgbaBufferToGrayscale(uint8_t *rgba, uint32_t width, uint32_t height, PariGpuBuffer gpu_in_buf,
                                              PariGpuBuffer gpu_out_buf, uint8_t *gray)
{
    uint64_t start = currentTime();

    // Get handles to input and output image pointers
    thrust::device_vector<uint8_t> *input_ptr = (thrust::device_vector<uint8_t>*)(gpu_in_buf[0]);
    thrust::device_vector<uint8_t> *output_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[0]);

    // Upload RGBA buffer to GPU
    thrust::copy(rgba, rgba + (width * height * 4), input_ptr->begin());

    // Convert RGBA buffer to Grayscale buffer (one thread per pixel)
    thrust::counting_iterator<size_t> it(0);
    thrust::for_each_n(thrust::device, it, width * height, PariGrayscaleFunctor(*input_ptr, *output_ptr));

    // Copy image data back to host
    thrust::copy(output_ptr->begin(), output_ptr->begin() + (width * height), gray);

    uint64_t end = currentTime();
    printf("PARI> pariRgbaBufferToGrayscale (%dx%d): %.6lf\n", width, height, (double)(end - start) / 1000000.0);
}

PARI_DLLEXPORT void pariRgbaBufferToDxt1(uint8_t *rgba, uint32_t width, uint32_t height, PariGpuBuffer gpu_in_buf,
                                         PariGpuBuffer gpu_out_buf,uint8_t *dxt1)
{
    uint64_t start = currentTime();

    // Get handles to input and output image pointers
    thrust::device_vector<uint8_t> *input_ptr = (thrust::device_vector<uint8_t>*)(gpu_in_buf[0]);
    thrust::device_vector<uint8_t> *output_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[0]);

    // Upload RGBA buffer to GPU
    thrust::copy(rgba, rgba + (width * height * 4), input_ptr->begin());

    // Convert RGBA buffer to DXT1 buffer (one thread per 4x4 tile)
    const int k = 16;                        // pixels per tile
    const int n = (width * height) / k;      // number of tiles
    thrust::counting_iterator<size_t> it(0);
    thrust::for_each_n(thrust::device, it, n, PariDxt1Functor(*input_ptr, *output_ptr, width));

    // Copy image data back to host
    thrust::copy(output_ptr->begin(), output_ptr->begin() + (width * height / 2), dxt1);

    uint64_t end = currentTime();
    printf("PARI> pariRgbaBufferToDxt1 (%dx%d): %.6lf\n", width, height, (double)(end - start) / 1000000.0);
}

PARI_DLLEXPORT void pariRgbaDepthBufferToActivePixel(uint8_t *rgba, float *depth, uint32_t width, uint32_t height,
                                                     PariGpuBuffer gpu_rgba_in_buf, PariGpuBuffer gpu_depth_in_buf,
                                                     PariGpuBuffer gpu_out_buf, uint8_t *active_pixel, uint32_t *active_pixel_size)
{
    uint64_t start = currentTime();

    // Get handles to input and output image pointers
    thrust::device_vector<uint8_t> *input_rgba_ptr = (thrust::device_vector<uint8_t>*)(gpu_rgba_in_buf[0]);
    thrust::device_vector<float> *input_depth_ptr = (thrust::device_vector<float>*)(gpu_depth_in_buf[0]);
    thrust::device_vector<uint8_t> *new_run_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[0]);
    thrust::device_vector<uint8_t> *is_active_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[1]);
    thrust::device_vector<uint32_t> *run_id_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[2]);
    thrust::device_vector<uint32_t> *run_counts_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[3]);
    thrust::device_vector<uint32_t> *active_idx_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[4]);
    thrust::device_vector<uint8_t> *output_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[5]);
    thrust::device_vector<uint32_t> *output_size_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[6]);

    // Upload RGBA and Depth buffers to GPU
    thrust::copy(rgba, rgba + (width * height * 4), input_rgba_ptr->begin());
    thrust::copy(depth, depth + (width * height), input_depth_ptr->begin());

    uint64_t start_compute = currentTime();

    // Convert RGBA and Depth buffers to Active Pixel buffer
    thrust::counting_iterator<size_t> it(0);
    typecast<uint8_t, uint32_t> ubyteToUint;
    thrust::plus<uint32_t> uintSum;
    //   - whether or not each pixel starts a new run (0 or 1) and whether or not each pixel is active (0 or 1)
    thrust::for_each_n(thrust::device, it, width * height, PariActivePixelFunctor(width, height, *input_rgba_ptr,
                       *input_depth_ptr, *new_run_ptr, *is_active_ptr));
    
    //   - id for each run
    thrust::transform_inclusive_scan(thrust::device, new_run_ptr->begin(), new_run_ptr->end(), run_id_ptr->begin(),
                                     ubyteToUint, uintSum);
    
    //   - number of pixels in each run
    thrust::reduce_by_key(thrust::device, run_id_ptr->begin(), run_id_ptr->end(), thrust::make_constant_iterator(1),
                          thrust::discard_iterator<uint32_t>(), run_counts_ptr->begin());
    
    //   - number of active pixels prior to each pixel
    thrust::transform_exclusive_scan(thrust::device, is_active_ptr->begin(), is_active_ptr->end(), active_idx_ptr->begin(),
                                     ubyteToUint, 0, uintSum);
    
    //   -  finalize compressed active pixel image
    thrust::for_each_n(thrust::device, it, width * height, PariActivePixelFinalizeFunctor(width, height, *input_rgba_ptr,
                       *input_depth_ptr, *is_active_ptr, *new_run_ptr, *run_id_ptr, *run_counts_ptr, *active_idx_ptr,
                       *output_ptr, *output_size_ptr));

    uint64_t end_compute = currentTime();

    // Copy image data back to host
    thrust::copy(output_size_ptr->begin(), output_size_ptr->end(), active_pixel_size);
    thrust::copy(output_ptr->begin(), output_ptr->begin() + (*active_pixel_size), active_pixel);

    uint64_t end = currentTime();
    printf("PARI> pariRgbaDepthBufferToActivePixel (%dx%d): %.6lf (%.6lf compute)\n", width, height, (double)(end - start) / 1000000.0, (double)(end_compute - start_compute) / 1000000.0);
}

PARI_DLLEXPORT double pariGetTime(PariEnum time)
{
    double elapsed = 0.0;
    switch (time)
    {
        case PARI_TIME_COMPUTE:
            elapsed = _compute_time;
            break;
        case PARI_TIME_MEMORY_TRANSFER:
            elapsed = _mem_transfer_time;
            break;
        case PARI_TIME_TOTAL:
            elapsed = _total_time;
            break;
    }
    return elapsed;
}

// OpenGL - PARI functions
PARI_DLLEXPORT PariCGResource pariRegisterImage(uint32_t texture, PariCGResourceDescription *resrc_description_ptr)
{
    struct hipGraphicsResource *cuda_resource;
    struct hipResourceDesc **description_ptr = (struct hipResourceDesc **)resrc_description_ptr;
    
    // NOTE: GL_DEPTH_COMPONENT not supported - only the following:
    //  - GL_RED, GL_RG, GL_RGBA, GL_LUMINANCE, GL_ALPHA, GL_LUMINANCE_ALPHA, GL_INTENSITY
    //  - {GL_R, GL_RG, GL_RGBA} X {8, 16, 16F, 32F, 8UI, 16UI, 32UI, 8I, 16I, 32I}
    //  - {GL_LUMINANCE, GL_ALPHA, GL_LUMINANCE_ALPHA, GL_INTENSITY} X {8, 16, 16F_ARB, 32F_ARB, 8UI_EXT, 16UI_EXT, 32UI_EXT, 8I_EXT, 16I_EXT, 32I_EXT}
    hipError_t err = hipGraphicsGLRegisterImage(&cuda_resource, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    if (err != hipSuccess)
    {
        fprintf(stderr, "PARI> PariCGResource: hipGraphicsGLRegisterImage - %s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
    }

    *description_ptr = new struct hipResourceDesc();
    memset(*description_ptr, 0, sizeof(struct hipResourceDesc));
    (*description_ptr)->resType = hipResourceTypeArray;
    
    return (PariCGResource)cuda_resource;
}

PARI_DLLEXPORT void pariUnregisterImage(PariCGResource resrc, PariCGResourceDescription resrc_description)
{
    struct hipGraphicsResource *cuda_resource = (struct hipGraphicsResource *)resrc;
    struct hipResourceDesc *description = (struct hipResourceDesc *)resrc_description;
    
    delete description;
    hipGraphicsUnregisterResource(cuda_resource);
}

PARI_DLLEXPORT void pariGetRgbaTextureAsGrayscale(PariCGResource cg_resource, PariCGResourceDescription resrc_description,
                                                  PariGpuBuffer gpu_out_buf, uint32_t width, uint32_t height, uint8_t *gray)
{
    glFinish(); // wait for OpenGL commands to finish and GPU to become available

    uint64_t start = currentTime();
    
    hipArray *array;
    hipSurfaceObject_t target;

    // Get handles to output image pointer as well as cuda resource and its description
    thrust::device_vector<uint8_t> *output_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[0]);
    struct hipGraphicsResource *cuda_resource = (struct hipGraphicsResource *)cg_resource;
    struct hipResourceDesc description = *(struct hipResourceDesc *)resrc_description;

    // Enable CUDA to access OpenGL texture
    hipGraphicsMapResources(1, &cuda_resource, 0);
    hipGraphicsSubResourceGetMappedArray(&array, cuda_resource, 0, 0);
    description.res.array.array = array;
    hipCreateSurfaceObject(&target, &description);
    
    // Convert RGBA texture to Grayscale buffer
    uint64_t start_compute = currentTime();
    thrust::counting_iterator<size_t> it(0);
    thrust::for_each_n(thrust::device, it, width * height, PariCGGrayscaleFunctor(target, *output_ptr, width, height));
    hipDeviceSynchronize();
    uint64_t end_compute = currentTime();

    // Copy image data back to host
    uint64_t start_mem_transfer = currentTime();
    thrust::copy(output_ptr->begin(), output_ptr->begin() + (width * height), gray);
    uint64_t end_mem_transfer = currentTime();

    // Release texture for use by OpenGL again
    hipDestroySurfaceObject(target);
    hipGraphicsUnmapResources(1, &cuda_resource, 0);

    uint64_t end = currentTime();

    _compute_time = (double)(end_compute - start_compute) / 1000000.0;
    _mem_transfer_time = (double)(end_mem_transfer - start_mem_transfer) / 1000000.0;
    _total_time = (double)(end - start) / 1000000.0;
}

PARI_DLLEXPORT void pariGetRgbaTextureAsDxt1(PariCGResource cg_resource, PariCGResourceDescription resrc_description,
                                             PariGpuBuffer gpu_out_buf, uint32_t width, uint32_t height, uint8_t *dxt1)
{
    glFinish(); // wait for OpenGL commands to finish and GPU to become available

    uint64_t start = currentTime();
    
    hipArray *array;
    hipSurfaceObject_t target;

    // Get handles to output image pointer as well as cuda resource and its description
    thrust::device_vector<uint8_t> *output_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[0]);
    struct hipGraphicsResource *cuda_resource = (struct hipGraphicsResource *)cg_resource;
    struct hipResourceDesc description = *(struct hipResourceDesc *)resrc_description;

    // Enable CUDA to access OpenGL texture
    hipGraphicsMapResources(1, &cuda_resource, 0);
    hipGraphicsSubResourceGetMappedArray(&array, cuda_resource, 0, 0);
    description.res.array.array = array;
    hipCreateSurfaceObject(&target, &description);

    // Convert RGBA texture to DXT1 buffer
    uint64_t start_compute = currentTime();
    const int k = 16;                        // pixels per tile
    const int n = (width * height) / k;      // number of tiles
    thrust::counting_iterator<size_t> it(0);
    thrust::for_each_n(thrust::device, it, n, PariCGDxt1Functor(target, *output_ptr, width, height));
    hipDeviceSynchronize();
    uint64_t end_compute = currentTime();

    // Copy image data back to host
    uint64_t start_mem_transfer = currentTime();
    thrust::copy(output_ptr->begin(), output_ptr->begin() + (width * height / 2), dxt1);
    uint64_t end_mem_transfer = currentTime();

    // Release texture for use by OpenGL again
    hipDestroySurfaceObject(target);
    hipGraphicsUnmapResources(1, &cuda_resource, 0);

    uint64_t end = currentTime();
    
    _compute_time = (double)(end_compute - start_compute) / 1000000.0;
    _mem_transfer_time = (double)(end_mem_transfer - start_mem_transfer) / 1000000.0;
    _total_time = (double)(end - start) / 1000000.0;
}

PARI_DLLEXPORT void pariGetRgbaDepthTextureAsActivePixel(PariCGResource cg_resource_color, PariCGResourceDescription resrc_description_color,
                                                         PariCGResource cg_resource_depth, PariCGResourceDescription resrc_description_depth,
                                                         PariGpuBuffer gpu_out_buf, uint32_t width, uint32_t height, uint8_t *active_pixel,
                                                         uint32_t *active_pixel_size)
{
    glFinish(); // wait for OpenGL commands to finish and GPU to become available
    //hipDeviceSynchronize();

    uint64_t start = currentTime();

    hipArray *array_color;
    hipArray *array_depth;
    hipSurfaceObject_t target_color;
    hipSurfaceObject_t target_depth;

    // Get handles to output image pointers as well as cuda resources and their descriptions
    thrust::device_vector<uint8_t> *new_run_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[0]);
    thrust::device_vector<uint8_t> *is_active_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[1]);
    thrust::device_vector<uint32_t> *run_id_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[2]);
    thrust::device_vector<uint32_t> *run_counts_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[3]);
    thrust::device_vector<uint32_t> *active_idx_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[4]);
    thrust::device_vector<uint8_t> *output_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[5]);
    thrust::device_vector<uint32_t> *output_size_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[6]);
    struct hipGraphicsResource *cuda_resource_color = (struct hipGraphicsResource *)cg_resource_color;
    struct hipGraphicsResource *cuda_resource_depth = (struct hipGraphicsResource *)cg_resource_depth;
    struct hipResourceDesc description_color = *(struct hipResourceDesc *)resrc_description_color;
    struct hipResourceDesc description_depth = *(struct hipResourceDesc *)resrc_description_depth;

    // Enable CUDA to access OpenGL texture
    hipGraphicsMapResources(1, &cuda_resource_color, 0);
    hipGraphicsMapResources(1, &cuda_resource_depth, 0);
    hipGraphicsSubResourceGetMappedArray(&array_color, cuda_resource_color, 0, 0);
    hipGraphicsSubResourceGetMappedArray(&array_depth, cuda_resource_depth, 0, 0);
    description_color.res.array.array = array_color;
    description_depth.res.array.array = array_depth;
    hipCreateSurfaceObject(&target_color, &description_color);
    hipCreateSurfaceObject(&target_depth, &description_depth);

    // Convert RGBA and Depth buffers to Active Pixel buffer
    uint64_t start_compute = currentTime();
    thrust::counting_iterator<size_t> it(0);
    typecast<uint8_t, uint32_t> ubyteToUint;
    thrust::plus<uint32_t> uintSum;
    //   - whether or not each pixel starts a new run (0 or 1) and whether or not each pixel is active (0 or 1)
    thrust::for_each_n(thrust::device, it, width * height, PariCGActivePixelFunctor(target_depth, *new_run_ptr,
                       *is_active_ptr, width, height));
    //   - id for each run
    thrust::transform_inclusive_scan(thrust::device, new_run_ptr->begin(), new_run_ptr->end(), run_id_ptr->begin(),
                                     ubyteToUint, uintSum);
    //   - number of pixels in each run
    thrust::reduce_by_key(thrust::device, run_id_ptr->begin(), run_id_ptr->end(), thrust::make_constant_iterator(1),
                          thrust::discard_iterator<uint32_t>(), run_counts_ptr->begin());
    //   - number of active pixels prior to each pixel
    thrust::transform_exclusive_scan(thrust::device, is_active_ptr->begin(), is_active_ptr->end(), active_idx_ptr->begin(),
                                     ubyteToUint, 0, uintSum);
    //   -  finalize compressed active pixel image
    thrust::for_each_n(thrust::device, it, width * height, PariCGActivePixelFinalizeFunctor(target_color, target_depth,
                       *is_active_ptr, *new_run_ptr, *run_id_ptr, *run_counts_ptr, *active_idx_ptr, *output_ptr,
                       *output_size_ptr, width, height));
    hipDeviceSynchronize();
    uint64_t end_compute = currentTime();

    // Copy image data back to host
    uint64_t start_mem_transfer = currentTime();
    thrust::copy(output_size_ptr->begin(), output_size_ptr->end(), active_pixel_size);
    thrust::copy(output_ptr->begin(), output_ptr->begin() + (*active_pixel_size), active_pixel);
    uint64_t end_mem_transfer = currentTime();

    uint64_t end = currentTime();
    
    _compute_time = (double)(end_compute - start_compute) / 1000000.0;
    _mem_transfer_time = (double)(end_mem_transfer - start_mem_transfer) / 1000000.0;
    _total_time = (double)(end - start) / 1000000.0;
}

PARI_DLLEXPORT void pariGetSubRgbaDepthTextureAsActivePixel(PariCGResource cg_resource_color, PariCGResourceDescription resrc_description_color,
                                                            PariCGResource cg_resource_depth, PariCGResourceDescription resrc_description_depth,
                                                            PariGpuBuffer gpu_out_buf, uint32_t texture_width, uint32_t texture_height,
                                                            int32_t *texture_viewport, uint32_t ap_width, uint32_t ap_height,
                                                            int32_t *ap_viewport, uint8_t *active_pixel, uint32_t *active_pixel_size)
{
    glFinish(); // wait for OpenGL commands to finish and GPU to become available
    //hipDeviceSynchronize();

    uint64_t start = currentTime();

    hipArray *array_color;
    hipArray *array_depth;
    hipSurfaceObject_t target_color;
    hipSurfaceObject_t target_depth;

    // Get handles to output image pointers as well as cuda resources and their descriptions
    thrust::device_vector<uint8_t> *new_run_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[0]);
    thrust::device_vector<uint8_t> *is_active_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[1]);
    thrust::device_vector<uint32_t> *run_id_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[2]);
    thrust::device_vector<uint32_t> *run_counts_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[3]);
    thrust::device_vector<uint32_t> *active_idx_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[4]);
    thrust::device_vector<uint8_t> *output_ptr = (thrust::device_vector<uint8_t>*)(gpu_out_buf[5]);
    thrust::device_vector<uint32_t> *output_size_ptr = (thrust::device_vector<uint32_t>*)(gpu_out_buf[6]);
    struct hipGraphicsResource *cuda_resource_color = (struct hipGraphicsResource *)cg_resource_color;
    struct hipGraphicsResource *cuda_resource_depth = (struct hipGraphicsResource *)cg_resource_depth;
    struct hipResourceDesc description_color = *(struct hipResourceDesc *)resrc_description_color;
    struct hipResourceDesc description_depth = *(struct hipResourceDesc *)resrc_description_depth;

    // Enable CUDA to access OpenGL texture
    hipGraphicsMapResources(1, &cuda_resource_color, 0);
    hipGraphicsMapResources(1, &cuda_resource_depth, 0);
    hipGraphicsSubResourceGetMappedArray(&array_color, cuda_resource_color, 0, 0);
    hipGraphicsSubResourceGetMappedArray(&array_depth, cuda_resource_depth, 0, 0);
    description_color.res.array.array = array_color;
    description_depth.res.array.array = array_depth;
    hipCreateSurfaceObject(&target_color, &description_color);
    hipCreateSurfaceObject(&target_depth, &description_depth);

    // Convert RGBA and Depth buffers to Active Pixel buffer
    uint64_t start_compute = currentTime();
    thrust::counting_iterator<size_t> it(0);
    typecast<uint8_t, uint32_t> ubyteToUint;
    thrust::plus<uint32_t> uintSum;
    uint32_t size = ap_width * ap_height;
    //   - whether or not each pixel starts a new run (0 or 1) and whether or not each pixel is active (0 or 1)
    thrust::for_each_n(thrust::device, it, size, PariCGSubActivePixelFunctor(target_depth, *new_run_ptr,
                       *is_active_ptr, texture_width, texture_height, texture_viewport, ap_width, ap_height, ap_viewport));
    //   - id for each run
    thrust::transform_inclusive_scan(thrust::device, new_run_ptr->begin(), new_run_ptr->begin() + size, run_id_ptr->begin(),
                                     ubyteToUint, uintSum);
    //   - number of pixels in each run
    thrust::reduce_by_key(thrust::device, run_id_ptr->begin(), run_id_ptr->begin() + size, thrust::make_constant_iterator(1),
                          thrust::discard_iterator<uint32_t>(), run_counts_ptr->begin());
    //   - number of active pixels prior to each pixel
    thrust::transform_exclusive_scan(thrust::device, is_active_ptr->begin(), is_active_ptr->begin() + size, active_idx_ptr->begin(),
                                     ubyteToUint, 0, uintSum);
    //   -  finalize compressed active pixel image
    thrust::for_each_n(thrust::device, it, size, PariCGSubActivePixelFinalizeFunctor(target_color, target_depth,
                       *is_active_ptr, *new_run_ptr, *run_id_ptr, *run_counts_ptr, *active_idx_ptr, *output_ptr,
                       *output_size_ptr, texture_width, texture_height, texture_viewport, ap_width, ap_height, ap_viewport));
    hipDeviceSynchronize();
    uint64_t end_compute = currentTime();

    // Copy image data back to host
    uint64_t start_mem_transfer = currentTime();
    thrust::copy(output_size_ptr->begin(), output_size_ptr->end(), active_pixel_size);
    thrust::copy(output_ptr->begin(), output_ptr->begin() + (*active_pixel_size), active_pixel);
    uint64_t end_mem_transfer = currentTime();

    uint64_t end = currentTime();
    
    _compute_time = (double)(end_compute - start_compute) / 1000000.0;
    _mem_transfer_time = (double)(end_mem_transfer - start_mem_transfer) / 1000000.0;
    _total_time = (double)(end - start) / 1000000.0;
}


// Internal functions
void extractTile4x4(uint32_t offset, const uint8_t *pixels, int width, uint8_t out_tile[64])
{
    int i, j;
    for (j = 0; j < 4; j++)
    {
        for (i = 0; i < 16; i++)
        {
            out_tile[j * 16 + i] = pixels[offset + i];
        }
        offset += width * 4;
    }
}

void getMinMaxColors(uint8_t tile[64], uint8_t color_min[3], uint8_t color_max[3])
{
    uint8_t inset[3];
    memset(color_min, 255, 3);
    memset(color_max, 0, 3);
    
    int i;
    for (i = 0; i < 16; i++)
    {
        color_min[0] = min(color_min[0], tile[i * 4 + 0]);
        color_min[1] = min(color_min[1], tile[i * 4 + 1]);
        color_min[2] = min(color_min[2], tile[i * 4 + 2]);
        color_max[0] = max(color_max[0], tile[i * 4 + 0]);
        color_max[1] = max(color_max[1], tile[i * 4 + 1]);
        color_max[2] = max(color_max[2], tile[i * 4 + 2]);
    }
    
    inset[0] = (color_max[0] - color_min[0]) >> 4;
    inset[1] = (color_max[1] - color_min[1]) >> 4;
    inset[2] = (color_max[2] - color_min[2]) >> 4;
    
    color_min[0] = min(color_min[0] + inset[0], 255);
    color_min[1] = min(color_min[1] + inset[1], 255);
    color_min[2] = min(color_min[2] + inset[2], 255);
    color_max[0] = max(color_max[0] - inset[0], 0);
    color_max[1] = max(color_max[1] - inset[1], 0);
    color_max[2] = max(color_max[2] - inset[2], 0);
}

uint16_t colorTo565(uint8_t color[3])
{
    return ((color[0] >> 3) << 11) | ((color[1] >> 2) << 5) | (color[2] >> 3);
}

uint32_t colorDistance(uint8_t tile[64], int t_offset, uint8_t colors[16], int c_offset)
{
    int dx = tile[t_offset + 0] - colors[c_offset + 0];
    int dy = tile[t_offset + 1] - colors[c_offset + 1];
    int dz = tile[t_offset + 2] - colors[c_offset + 2];
    
    return (dx*dx) + (dy*dy) + (dz*dz);
}

uint32_t colorIndices(uint8_t tile[64], uint8_t color_min[3], uint8_t color_max[3])
{
    uint8_t colors[16];
    uint8_t indices[16];
    int i, j;
    uint8_t C565_5_MASK = 0xF8;   // 0xFF minus last three bits
    uint8_t C565_6_MASK = 0xFC;   // 0xFF minus last two bits
    
    colors[0] = (color_max[0] & C565_5_MASK) | (color_max[0] >> 5);
    colors[1] = (color_max[1] & C565_6_MASK) | (color_max[1] >> 6);
    colors[2] = (color_max[2] & C565_5_MASK) | (color_max[2] >> 5);
    colors[4] = (color_min[0] & C565_5_MASK) | (color_min[0] >> 5);
    colors[5] = (color_min[1] & C565_6_MASK) | (color_min[1] >> 6);
    colors[6] = (color_min[2] & C565_5_MASK) | (color_min[2] >> 5);
    colors[8] = (2 * colors[0] + colors[4]) / 3;
    colors[9] = (2 * colors[1] + colors[5]) / 3;
    colors[10] = (2 * colors[2] + colors[6]) / 3;
    colors[12] = (colors[0] + 2 * colors[4]) / 3;
    colors[13] = (colors[1] + 2 * colors[5]) / 3;
    colors[14] = (colors[2] + 2 * colors[6]) / 3;
    
    uint32_t dist, min_dist;
    for (i = 0; i < 16; i++)
    {
        min_dist = 195076;  // 255 * 255 * 3 + 1
        for (j = 0; j < 4; j++)
        {
            dist = colorDistance(tile, i * 4, colors, j * 4);
            if (dist < min_dist)
            {
                min_dist = dist;
                indices[i] = j;
            }
        }
    }
    
    uint32_t result = 0;
    for (i = 0; i < 16; i++)
    {
        result |= indices[i] << (i * 2);
    }
    return result;
}

void writeUint16(uint8_t *buffer, uint32_t offset, uint16_t value)
{
   buffer[offset + 0] = value & 0xFF;
   buffer[offset + 1] = (value >> 8) & 0xFF;
}

void writeUint32(uint8_t *buffer, uint32_t offset, uint32_t value)
{
    buffer[offset + 0] = value & 0xFF;
    buffer[offset + 1] = (value >> 8) & 0xFF;
    buffer[offset + 2] = (value >> 16) & 0xFF;
    buffer[offset + 3] = (value >> 24) & 0xFF;
}

__device__ void extractCGTile4x4(uint32_t offset_x, uint32_t offset_y, const hipSurfaceObject_t pixels, uint8_t out_tile[64])
{
    int i, j;
    for (j = 0; j < 4; j++)
    {
        for (i = 0; i < 4; i++)
        {
            uchar4 color;
            surf2Dread(&color, pixels, 4 * (offset_x + i), offset_y + j);
            out_tile[j * 16 + 4 * i + 0] = color.x;
            out_tile[j * 16 + 4 * i + 1] = color.y;
            out_tile[j * 16 + 4 * i + 2] = color.z;
            out_tile[j * 16 + 4 * i + 3] = color.w;
        }
    }
}

// --------------------------------------------------------------- //

static uint64_t currentTime()
{
    struct timespec ts;
    timespec_get(&ts, TIME_UTC);
    return (ts.tv_sec * 1000000ull) + (ts.tv_nsec / 1000ull);
}
